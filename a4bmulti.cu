#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2 /* Number of rows of first matrix */
#define col1 3 /* Number of columns of first matrix */
#define row2 3 /* Number of rows of second matrix */
#define col2 2 /* Number of columns of second matrix */

__global__ void matproduct(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k;
  
n[col2*y+x]=0;
for(k=0;k<col1;k++)
   {
    n[col2*y+x]=n[col2*y+x]+l[col1*y+k]*m[col2*k+x];
   }
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

    printf("\n Enter elements of first matrix of size 2*3\n");
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col1;j++)
            {
                scanf("%d",&a[i][j]);
            }
    }
    printf("\n Enter elements of second matrix of size 3*2\n");
        for(i=0;i<row2;i++)
        {
            for(j=0;j<col2;j++)
                {
                    scanf("%d",&b[i][j]);
                }
        }

    hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */

    matproduct<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nProduct of two matrices:\n ");
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col2;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
// #include <cuda_runtime.h>
// #include <iostream>

// __global__ void matmul(int* A, int* B, int* C, int N) {
//     int Row = blockIdx.y*blockDim.y+threadIdx.y;
//     int Col = blockIdx.x*blockDim.x+threadIdx.x;
//     if (Row < N && Col < N) {
//         int Pvalue = 0;
//         for (int k = 0; k < N; k++) {
//             Pvalue += A[Row*N+k] * B[k*N+Col];
//         }
//         C[Row*N+Col] = Pvalue;
//     }
// }

// int main() {
//     int N = 512;
//     int size = N * N * sizeof(int);
//     int* A, * B, * C;
//     int* dev_A, * dev_B, * dev_C;
//     cudaMallocHost(&A, size);
//     cudaMallocHost(&B, size);
//     cudaMallocHost(&C, size);
//     cudaMalloc(&dev_A, size);
//     cudaMalloc(&dev_B, size);
//     cudaMalloc(&dev_C, size);

//     // Initialize matrices A and B
//     for (int i = 0; i < N; i++) {
//         for (int j = 0; j < N; j++) {
//             A[i*N+j] = i*N+j;
//             B[i*N+j] = j*N+i;
//         }
//     }

//     cudaMemcpy(dev_A, A, size, cudaMemcpyHostToDevice);
//     cudaMemcpy(dev_B, B, size, cudaMemcpyHostToDevice);

//     dim3 dimBlock(16, 16);
//     dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

//     matmul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);

//     cudaMemcpy(C, dev_C, size, cudaMemcpyDeviceToHost);

//     // Print the result
//     for (int i = 0; i < 10; i++) {
//         for (int j = 0; j < 10; j++) {
//             std::cout << C[i*N+j] << " ";
//         }
//         std::cout << std::endl;
//     }

//     // Free memory
//     cudaFree(dev_A);
//     cudaFree(dev_B);
//     cudaFree(dev_C);
//     cudaFreeHost(A);
//     cudaFreeHost(B);
//     cudaFreeHost(C);

//     return 0;
// }
